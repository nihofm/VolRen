#include "hip/hip_runtime.h"
//
// Copyright (c) 2021, NVIDIA CORPORATION. All rights reserved.
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions
// are met:
//  * Redistributions of source code must retain the above copyright
//    notice, this list of conditions and the following disclaimer.
//  * Redistributions in binary form must reproduce the above copyright
//    notice, this list of conditions and the following disclaimer in the
//    documentation and/or other materials provided with the distribution.
//  * Neither the name of NVIDIA CORPORATION nor the names of its
//    contributors may be used to endorse or promote products derived
//    from this software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
// EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
// PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
// CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
// EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
// PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
// PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
// OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
// (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
// OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
//

#include "common.cuh"
#include "device_helpers.cuh"

extern "C" {
__constant__ Params params;
}

extern "C" __global__ void __raygen__draw_solid_color() {
    uint3 launch_index = optixGetLaunchIndex();
    size_t idx = launch_index.y * params.resolution.x + launch_index.x;
    RayGenData* rtData = (RayGenData*)optixGetSbtDataPointer();
    params.image[idx] = make_float4(rtData->r, rtData->g, rtData->b, 0);
    const float3 dir = view_dir(params.cam_dir, params.cam_fov, make_int2(launch_index.x, launch_index.y), make_int2(params.resolution.x, params.resolution.y));
    params.image[idx] = fabs(make_float4(dir.x, dir.y, dir.z, 0));
}
